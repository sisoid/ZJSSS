
#include <hip/hip_runtime.h>
#include <cstdio>
#include <ctime>
#include <cstdlib>
#include <omp.h>
#include <iostream>

#define COUNTERS 66
#define C_SIZE 64
#define C_STOP 65 // == C_SIZE+1
#define N 4224 // == COUNTERS*C_SIZE
#define N2 17842176 // == N*N

#define CUDA_ERROR_CHECK

#define cudaSafeCall(error) __cudaSafeCall(error, __FILE__, __LINE__)
#define cudaCheckErrors() __cudaCheckErrors(__FILE__, __LINE__)

inline void __cudaSafeCall(hipError_t error, const char *file, const int line) {
#ifdef CUDA_ERROR_CHECK
    if (error != hipSuccess) {
        std::cout << "error: CudaSafeCall() failed at " << file << ":" << line
                  << " with \"" << hipGetErrorString(error) << "\""
                  << std::endl;
        exit(-1);
    }
#endif
}

inline void __cudaCheckErrors(const char *file, const int line) {
#ifdef CUDA_ERROR_CHECK
    __cudaSafeCall(hipGetLastError(), file, line);
#endif
}



__device__ inline int uniq(const int* M, int i, int* counters) {
    for (int j = 1; j <= i - 1; j++) {
        int a = (j - 1) * C_SIZE + counters[j-1];
        int b = (i - 1) * C_SIZE + counters[i-1];
        if (M[(a - 1) + N * (b - 1)] == 0)
            return 1;
    }
    return 0;
}

__global__ void searcher(const int* M, int* res, size_t* itersNum) {
    int partNumber = threadIdx.x + blockIdx.x * blockDim.x;
    // initialize counters vector
    int counters[COUNTERS];
    for (int i = 0; i < COUNTERS; i++)
        counters[i] = 1;

    // go to selected part
    counters[0] = 25;
    counters[1] = 5;
    counters[2] = 1;
    counters[3] = 3;
    counters[4] = 4;
    counters[5] = 7;
    counters[6] = 9;
    counters[7] = 2;
    counters[8] = 10;
    counters[9] = 8;
    counters[10] = (partNumber - 1) / 64 + 1;
    counters[11] = (partNumber - 1) % 64 + 1;

    size_t iter = 0;
    size_t current = 1;
    while (1) {
        iter++;

        // stop if search in the selected part is finished
        if (counters[10] != (partNumber - 1) / 64 + 1 || counters[11] != (partNumber - 1) % 64 + 1) {
            for (int i = 0; i < COUNTERS; i++)
                res[partNumber * COUNTERS + i] = -1;
            itersNum[partNumber] = iter;
            break;
        }

        // first subspace is always good
        if (current == 1)
            current = 2;

        // print intermediate state
        // if (current == 13 && iter > 1000) {
        //     fprintf(f, "Current state of part number %d:", partNumber);
        //     for (int i = 0; i < COUNTERS; i++)
        //         fprintf(f, " %d", counters[i]);
        //     fprintf(f, "\nNumber of iterations: %f\n\n", iter);
        //     fflush(f);
        // }

        for (int i = current; i <= COUNTERS; i++) {
            if (uniq(M, i, counters) == 1) {
                counters[i-1]++;
                current = i;
                while (counters[current-1] == C_STOP) {
                    counters[current - 1] = 1;
                    counters[current - 2] = counters[current - 2] + 1;
                    current--;
                }
                break;
            }
        }

        if (current == COUNTERS && uniq(M, current, counters) == 0) {
            for (int i = 0; i < COUNTERS; i++)
                res[partNumber * COUNTERS + i] = counters[i];
            itersNum[partNumber] = iter;
            break;
        }
    }
}

int main() {
    int *M = new int[N2];
    // read intersection matrix from file
    FILE *f = fopen("input.tsv", "r");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            fscanf(f, "%d", &M[i + N * j]);
        }
    }
    fclose(f);

    int *d_M;
    hipMalloc((void **)&d_M, N2 * sizeof(int));
    hipMemcpy(d_M, M, N2 * sizeof(int), hipMemcpyHostToDevice);


    int *res = new int[COUNTERS * 4096];
    int *d_res;
    hipMalloc((void **)&d_res, COUNTERS * 4096 * sizeof(int));

    size_t *itersNum = new size_t[4096];
    memset(itersNum, 0, 4096 * sizeof(size_t));
    size_t *d_itersNum;
    hipMalloc((void **)&d_itersNum, 4096 * sizeof(size_t));
    hipMemcpy(d_itersNum, itersNum, 4096 * sizeof(size_t), hipMemcpyHostToDevice);

    // omp_set_num_threads(24);
    // #pragma omp parallel for
    // for (int i=0; i<4096; i++) {
    //     searcher(d_M, d_res, d_itersNum);
    // }

    searcher<<<32, 128>>>(d_M, d_res, d_itersNum);
    hipDeviceSynchronize();
    cudaCheckErrors();

    hipMemcpy(res, d_res, COUNTERS * 4096 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(itersNum, d_itersNum, 4096 * sizeof(size_t), hipMemcpyDeviceToHost);


    f = fopen("output.tsv", "w");
    for (int partNumber = 0; partNumber < 4096; partNumber++) {
        for (int i = 0; i < COUNTERS; i++) {
            fprintf(f, "%d\t", res[partNumber * COUNTERS + i]);
        }
        fprintf(f, "\n");
        printf("%zu ", itersNum[partNumber]);
    }
    fclose(f);

    hipFree(d_M);
    hipFree(d_res);
    hipFree(d_itersNum);
    delete [] M;
    delete [] res;
    delete [] itersNum;
    return 0;
}
